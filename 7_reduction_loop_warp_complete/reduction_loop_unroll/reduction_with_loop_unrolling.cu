#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "host_utils.h"
#include "cuda_utils.cuh"

#include <iostream>
#include <stdlib.h>

using namespace std;

/* //TODO
__global__ void ReduceSumGPU(int *input, int *sum, int size) {
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + tid;

	__syncthreads();
}
*/

__global__ void ReduceSumLoopUnroll(int *input, int *temp, int size) {
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x * 2;
	int index = block_offset + tid;

	int *i_data = input + block_offset;

	if ((index + blockDim.x) < size) {
		input[index] += input[index + blockDim.x];
	}
	__syncthreads();

	for (int offset = blockDim.x / 2; offset > 0; offset = offset / 2) {
		if (tid < offset) {
			i_data[tid] += i_data[tid + offset];
		}
		__syncthreads();
	}

	if (tid == 0) {
		temp[blockIdx.x] = i_data[0];
	}
}

int main() {
	cout << "\n-----Reduce Sum with Loop Unroll-----\n" << endl;
	float gpu_total_milliseconds = 0;
	float kernel_milliseconds = 0;
	clock_t cpu_start, cpu_end;
	hipEvent_t gpu_start, gpu_end, kernel_start, kernel_end;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_end);
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_end);

	int size = 1 << 20; // 1MB
	int block_size = 128;
	int byte_size = size * sizeof(int);

	int *host_input, *host_ref_unroll;
	host_input = (int*)malloc(byte_size);

	InitializeData(host_input, size, INIT_ONE);

	// Perform reduce sum on CPU
	cpu_start = clock();
	int cpu_result = ReductionSumCPU(host_input, size);
	cpu_end = clock();

	dim3 block(block_size);
	dim3 grid(ceil((size / block_size) / 2));

	printf("Kernel launch parameters -> grid: (%d,%d,%d), block: (%d,%d,%d) \n\n",
            grid.x, grid.y, grid.z, block.x, block.y, block.z);

	int temp_array_byte_size = sizeof(int)* grid.x;
	host_ref_unroll = (int*)malloc(temp_array_byte_size);

	int *device_input, *device_temp;
	GPUErrorCheck(hipEventRecord(gpu_start, 0));
	GPUErrorCheck(hipMalloc((void**)&device_input, byte_size));
	GPUErrorCheck(hipMemcpy(device_input, host_input, byte_size, hipMemcpyHostToDevice));
	GPUErrorCheck(hipMalloc((void**)&device_temp, temp_array_byte_size));
	GPUErrorCheck(hipMemset(device_temp, 0, temp_array_byte_size));

	// Perform reduce sum with loop unroll on GPU
	GPUErrorCheck(hipEventRecord(kernel_start, 0));
	ReduceSumLoopUnroll<<<grid, block>>>(device_input, device_temp, size);
	GPUErrorCheck(hipEventRecord(kernel_end, 0));
	GPUErrorCheck(hipEventSynchronize(kernel_end));

	GPUErrorCheck(hipDeviceSynchronize());
	GPUErrorCheck(hipMemcpy(host_ref_unroll, device_temp, temp_array_byte_size, hipMemcpyDeviceToHost));

	int gpu_result = 0;
	for (int i = 0; i < grid.x; i++) {
		gpu_result += host_ref_unroll[i];
	}
	GPUErrorCheck(hipEventRecord(gpu_end, 0));
	GPUErrorCheck(hipEventSynchronize(gpu_end));
	GPUErrorCheck(hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_end));
	GPUErrorCheck(hipEventElapsedTime(&gpu_total_milliseconds, gpu_start, gpu_end));

	CompareResults(gpu_result, cpu_result);

	printf("CPU execution time (Function Only): %4.6f milliseconds",
		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC) * 1000.0);
	printf("\nGPU Execution Time (Kernel Only): %4.6f milliseconds", kernel_milliseconds);
	printf("\nTotal GPU Execution Time (Malloc, Memcpy, Kernel): %4.6f milliseconds\n", gpu_total_milliseconds);

	GPUErrorCheck(hipFree(device_input));
	GPUErrorCheck(hipFree(device_temp));
	free(host_input);
	free(host_ref_unroll);

	GPUErrorCheck(hipDeviceReset());
	return 0;
}