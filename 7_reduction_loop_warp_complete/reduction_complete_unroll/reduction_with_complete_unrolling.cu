#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "host_utils.h"
#include "cuda_utils.cuh"

#include <iostream>
#include <stdlib.h>

using namespace std;

/* //TODO
__global__ void ReduceSumGPU(int *input, int *sum, int size) {
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + tid;

	__syncthreads();
}
*/

template<unsigned int block_size>
__global__ void ReduceSumCompleteUnroll(int *input, int *temp, int size) {
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x * 2;
	int index = block_offset + tid;

	int *i_data = input + block_offset;

	if ((index + blockDim.x) < size) {
		input[index] += input[index + blockDim.x];
	}
	__syncthreads();

	// Complete unrolling
	// Unrolling based on block size
	// In-place Reduction
	if (block_size >= 1024 && tid < 512)
		i_data[tid] += i_data[tid + 512];

	__syncthreads();

	if (block_size >= 512 && tid < 256)
		i_data[tid] += i_data[tid + 256];

	__syncthreads();

	if (block_size >= 256 && tid < 128)
		i_data[tid] += i_data[tid + 128];

	__syncthreads();

	if (block_size >= 128 && tid < 64)
		i_data[tid] += i_data[tid + 64];

	__syncthreads();

	// Warp unrolling
	if (tid < 32) {
		volatile int *vsmem = i_data;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0) {
		temp[blockIdx.x] = i_data[0];
	}
}

template<unsigned int block_size>
__global__ void ReduceSumCompleteUnrollSMem(int *input, int *temp, int size) {
	__shared__ int smem[block_size];
	unsigned int tid = threadIdx.x;

	// Global index
	// 4 blocks of input data processed at a time
	int block_offset = blockIdx.x * blockDim.x * 4;  
	unsigned int index = block_offset + tid;  

	int temp_sum = 0;

	// Unrolling 4 blocks
	if (index + 3 * blockDim.x <= size) { // Boundary check
		int val1 = input[index + 0 * blockDim.x];
		int val2 = input[index + 1 * blockDim.x];
		int val3 = input[index + 2 * blockDim.x];
		int val4 = input[index + 3 * blockDim.x];
		temp_sum = val1 + val2 + val3 + val4;
	}

	smem[tid] = temp_sum;

	__syncthreads();

	// Complete unrolling
	// Unrolling based on block size
	// In-place Reduction in Shared Memory
	if (block_size >= 1024 && tid < 512) {
		smem[tid] += smem[tid + 512];
	}
	__syncthreads();

	if (block_size >= 512 && tid < 256) {
		smem[tid] += smem[tid + 256];
	}
	__syncthreads();

	if (block_size >= 256 && tid < 128) {
		smem[tid] += smem[tid + 128];
	}
	__syncthreads();

	if (block_size >= 128 && tid < 64) {
		smem[tid] += smem[tid + 64];
	}
	__syncthreads();

	// Warp unrolling
	if (tid < 32) {
		volatile int * vsmem = smem;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0) {
		temp[blockIdx.x] = smem[0];
	}
}

int main() {
	float gpu_total_milliseconds = 0;
	float kernel_milliseconds = 0;
	clock_t cpu_start, cpu_end;
	hipEvent_t gpu_start, gpu_end, kernel_start, kernel_end;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_end);
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_end);

	int size = 1 << 20; // 1MB
	int block_size = 1024;
	int byte_size = size * sizeof(int);

	int *host_input, *host_ref_unroll;
	host_input = (int*)malloc(byte_size);

	InitializeData(host_input, size, INIT_ONE);

	// Perform reduce sum on CPU
	cout << "\n-----CPU: Reduce Sum-----\n" << endl;
	cpu_start = clock();
	int cpu_result = ReductionSumCPU(host_input, size);
	cpu_end = clock();

	printf("CPU execution time (Function Only): %4.6f milliseconds",
		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC) * 1000.0);

	int gpu_result = 0;
	dim3 block(block_size);
	dim3 grid(ceil((size / block_size) / 2));

	printf("\nKernel launch parameters -> grid: (%d,%d,%d), block: (%d,%d,%d) \n",
            grid.x, grid.y, grid.z, block.x, block.y, block.z);

	int temp_array_byte_size = sizeof(int) * grid.x;
	host_ref_unroll = (int*)malloc(temp_array_byte_size);

	int *device_input, *device_temp;
	GPUErrorCheck(hipEventRecord(gpu_start, 0));
	GPUErrorCheck(hipMalloc((void**)&device_input, byte_size));
	GPUErrorCheck(hipMemcpy(device_input, host_input, byte_size, hipMemcpyHostToDevice));
	GPUErrorCheck(hipMalloc((void**)&device_temp, temp_array_byte_size));
	GPUErrorCheck(hipMemset(device_temp, 0, temp_array_byte_size));

	// Perform reduce sum with complete unroll on GPU
	cout << "\n-----Device: Reduce Sum with Complete Unroll-----\n" << endl;
	GPUErrorCheck(hipEventRecord(kernel_start, 0));
	switch(block_size) {
		case 1024:
			ReduceSumCompleteUnroll<1024><<<grid, block>>>(device_input, device_temp, size);
			break;
		case 512:
			ReduceSumCompleteUnroll<512><<<grid, block>>>(device_input, device_temp, size);
			break;
		case 256:
			ReduceSumCompleteUnroll<256><<<grid, block>>>(device_input, device_temp, size);
			break;
		case 128:
			ReduceSumCompleteUnroll<128><<<grid, block>>>(device_input, device_temp, size);
			break;
	}
	GPUErrorCheck(hipEventRecord(kernel_end, 0));
	GPUErrorCheck(hipEventSynchronize(kernel_end));

	GPUErrorCheck(hipDeviceSynchronize());
	GPUErrorCheck(hipMemcpy(host_ref_unroll, device_temp, temp_array_byte_size, hipMemcpyDeviceToHost));

	for (int i = 0; i < grid.x; i++) {
		gpu_result += host_ref_unroll[i];
	}
	GPUErrorCheck(hipEventRecord(gpu_end, 0));
	GPUErrorCheck(hipEventSynchronize(gpu_end));
	GPUErrorCheck(hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_end));
	GPUErrorCheck(hipEventElapsedTime(&gpu_total_milliseconds, gpu_start, gpu_end));

	CompareResults(gpu_result, cpu_result);

	printf("GPU Execution Time (Kernel Only): %4.6f milliseconds", kernel_milliseconds);
	printf("\nTotal GPU Execution Time (Malloc, Memcpy, Kernel): %4.6f milliseconds\n", gpu_total_milliseconds);

	GPUErrorCheck(hipFree(device_temp));
	free(host_ref_unroll);

	grid.x = ceil((size / block_size) / 4);
	temp_array_byte_size = sizeof(int) * grid.x;
	host_ref_unroll = (int*)malloc(temp_array_byte_size);
	memset(host_ref_unroll, 0, temp_array_byte_size);
	GPUErrorCheck(hipMalloc((void**)&device_temp, temp_array_byte_size));
	GPUErrorCheck(hipMemset(device_temp, 0, temp_array_byte_size));

	cout << "\n-----Device: Reduce Sum using Shared Memory with Complete Unroll-----\n" << endl;
	GPUErrorCheck(hipEventRecord(kernel_start, 0));
	switch(block_size) {
		case 1024:
			ReduceSumCompleteUnrollSMem<1024><<<grid, block>>>(device_input, device_temp, size);
			break;
		case 512:
			ReduceSumCompleteUnrollSMem<512><<<grid, block>>>(device_input, device_temp, size);
			break;
		case 256:
			ReduceSumCompleteUnrollSMem<256><<<grid, block>>>(device_input, device_temp, size);
			break;
		case 128:
			ReduceSumCompleteUnrollSMem<128><<<grid, block>>>(device_input, device_temp, size);
			break;
	}
	GPUErrorCheck(hipEventRecord(kernel_end, 0));
	GPUErrorCheck(hipEventSynchronize(kernel_end));
	GPUErrorCheck(hipDeviceSynchronize());

	GPUErrorCheck(hipMemcpy(host_ref_unroll, device_temp, temp_array_byte_size, hipMemcpyDeviceToHost));

	gpu_result = 0;
	for (int i = 0; i < grid.x; i++) {
		gpu_result += host_ref_unroll[i];
	}
	CompareResults(gpu_result, cpu_result);

	GPUErrorCheck(hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_end));
	printf("GPU Execution Time (Kernel Only): %4.6f milliseconds", kernel_milliseconds);

	GPUErrorCheck(hipFree(device_temp));
	free(host_ref_unroll);

	GPUErrorCheck(hipFree(device_input));
	free(host_input);

	GPUErrorCheck(hipDeviceReset());
	return 0;
}