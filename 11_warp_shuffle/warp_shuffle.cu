#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "host_utils.h"
#include "cuda_utils.cuh"

#include <iostream>
#include <stdlib.h>

using namespace std;

#define MASK 0xffffffff
#define WARP_SIZE 32

__global__ void WarpShuffle(int *input, int *output, int srcLane, int width=WARP_SIZE) {
    auto in = input[threadIdx.x];
    auto out = __shfl_sync(MASK, in, srcLane, width);
    output[threadIdx.x] = out;
}

__global__ void WarpShuffleUP(int *input, int *output, unsigned int delta, int width=WARP_SIZE) {
    auto in = input[threadIdx.x];
    auto out = __shfl_up_sync(MASK, in, delta, width);
    output[threadIdx.x] = out;
}

__global__ void WarpShuffleDOWN(int *input, int *output, unsigned int delta, int width=WARP_SIZE) {
    auto in = input[threadIdx.x];
    auto out = __shfl_down_sync(MASK, in, delta, width);
    output[threadIdx.x] = out;
}

__global__ void WarpShuffleXOR(int *input, int *output, int laneMask, int width=WARP_SIZE) {
    auto in = input[threadIdx.x];
    auto out = __shfl_xor_sync(MASK, in, laneMask, width);
    output[threadIdx.x] = out;
}

__global__ void ReduceSumWarpShuffle(int *input, int *output, int size, int width=WARP_SIZE) {
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x * 2;
	int index = block_offset + tid;

	int *i_data = input + block_offset;

	if ((index + blockDim.x) < size) {
		input[index] += input[index + blockDim.x];
	}
	__syncthreads();

	// Loop unrolling
	for (int offset = blockDim.x / 2; offset >= 32; offset = offset / 2) {
		if (tid < offset) {
			i_data[tid] += i_data[tid + offset];
		}
		__syncthreads();
	}

    int sum = i_data[tid];
	// Warp unrolling
	if (tid < 32) {
        sum += __shfl_down_sync(MASK, sum, 16, width);
        sum += __shfl_down_sync(MASK, sum, 8, width);
        sum += __shfl_down_sync(MASK, sum, 4, width);
        sum += __shfl_down_sync(MASK, sum, 2, width);
        sum += __shfl_down_sync(MASK, sum, 1, width);
	}

	if (tid == 0) {
		output[blockIdx.x] = sum;
	}
}

int main() {
    int srcLane, delta, laneMask;
    int size = 64;
    int byte_size = size * sizeof(int);

    int *host_input, *host_output;
    host_input = (int*)malloc(byte_size);
    InitializeData(host_input, size, INIT_RANGE);
    cout << "\nInput Array:\n" << endl;
    PrintArray(host_input, size);

    auto nx = size;
    dim3 block(nx);
    dim3 grid(nx / block.x);

    int *device_input, *device_output;
    GPUErrorCheck(hipMalloc((void**)&device_input, byte_size));
    GPUErrorCheck(hipMemcpy(device_input, host_input, byte_size, hipMemcpyHostToDevice));

    cout << "\nWarp Shuffle:\n" << endl;
    srcLane = 4;
    GPUErrorCheck(hipMalloc((void**)&device_output, byte_size));
    WarpShuffle<<<grid, block>>>(device_input, device_output, srcLane);
    GPUErrorCheck(hipDeviceSynchronize());
    host_output = (int*)malloc(byte_size);
    GPUErrorCheck(hipMemcpy(host_output, device_output, byte_size, hipMemcpyDeviceToHost));
    PrintArray(host_output, size);
    GPUErrorCheck(hipFree(device_output));
    free(host_output);

    cout << "\nWarp Shuffle Up:\n" << endl;
    delta = 4;
    GPUErrorCheck(hipMalloc((void**)&device_output, byte_size));
    WarpShuffleUP<<<grid, block>>>(device_input, device_output, delta);
    GPUErrorCheck(hipDeviceSynchronize());
    host_output = (int*)malloc(byte_size);
    GPUErrorCheck(hipMemcpy(host_output, device_output, byte_size, hipMemcpyDeviceToHost));
    PrintArray(host_output, size);
    GPUErrorCheck(hipFree(device_output));
    free(host_output);

    cout << "\nWarp Shuffle Down:\n" << endl;
    delta = 4;
    GPUErrorCheck(hipMalloc((void**)&device_output, byte_size));
    WarpShuffleDOWN<<<grid, block>>>(device_input, device_output, delta);
    GPUErrorCheck(hipDeviceSynchronize());
    host_output = (int*)malloc(byte_size);
    GPUErrorCheck(hipMemcpy(host_output, device_output, byte_size, hipMemcpyDeviceToHost));
    PrintArray(host_output, size);
    GPUErrorCheck(hipFree(device_output));
    free(host_output);

    cout << "\nWarp Shuffle XOR:\n" << endl;
    laneMask = 1;
    GPUErrorCheck(hipMalloc((void**)&device_output, byte_size));
    WarpShuffleXOR<<<grid, block>>>(device_input, device_output, laneMask);
    GPUErrorCheck(hipDeviceSynchronize());
    host_output = (int*)malloc(byte_size);
    GPUErrorCheck(hipMemcpy(host_output, device_output, byte_size, hipMemcpyDeviceToHost));
    PrintArray(host_output, size);
    GPUErrorCheck(hipFree(device_output));
    free(host_output);

    cout << "\nReduce Sum with Warp Shuffle:\n" << endl;
    int cpu_result = ReductionSumCPU(host_input, size);

    int temp_array_byte_size = sizeof(int)* grid.x;
    GPUErrorCheck(hipMalloc((void**)&device_output, temp_array_byte_size));
    ReduceSumWarpShuffle<<<grid, block>>>(device_input, device_output, size);
    GPUErrorCheck(hipDeviceSynchronize());
    host_output = (int*)malloc(temp_array_byte_size);
    GPUErrorCheck(hipMemcpy(host_output, device_output, temp_array_byte_size, hipMemcpyDeviceToHost));

    int gpu_result = 0;
    for (int i = 0; i < grid.x; i++) {
        gpu_result += host_output[i];
    }
    CompareResults(gpu_result, cpu_result);
    GPUErrorCheck(hipFree(device_output));
    free(host_output);

    GPUErrorCheck(hipFree(device_input));
    free(host_input);

    GPUErrorCheck(hipDeviceReset());
    return 0;
}