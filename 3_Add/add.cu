#include "hip/hip_runtime.h"


#include <iostream>

__global__ void AddKernel(int *c, const int *a, const int *b) {
    /** 
     * Add two arrays.
    **/
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

void PrintInput(const int arr[], const int arraysize) {
    printf("[ ");
    for (int i = 0; i < arraysize; i++) {
        printf("%d, ", arr[i]);
    }
    printf("]");
}

void PrintOutput(int *arr, const int arraysize) {
    printf("[ ");
    for (int i = 0; i < arraysize; i++) {
        printf("%d, ", arr[i]);
    }
    printf("]");
}

int main() {
    const int arraysize = 5;
    const int a[arraysize] = { 0, 1, 2, 3, 4 };
    const int b[arraysize] = { 4, 3, 2, 1, 0 };

    // Transfer data from host to device
    int *device_a;
    hipMalloc((void**)&device_a, sizeof(int) * arraysize);
    hipMemcpy(device_a, a, sizeof(int) * arraysize, hipMemcpyHostToDevice);

    int *device_b;
    hipMalloc((void**)&device_b, sizeof(int) * arraysize);
    hipMemcpy(device_b, b, sizeof(int) * arraysize, hipMemcpyHostToDevice);

    int* device_c;
    hipMalloc((void**)&device_c, sizeof(int) * arraysize);

    // Set block and grid size
    int nx = 5;
    dim3 block(5);
    dim3 grid(nx / block.x);

    // Launch Add kernel
    printf("\nAdding two vectors...\n\n");
    PrintInput(a, arraysize);
    printf("\t + \t");
    PrintInput(b, arraysize);
    AddKernel<<<grid, block>>>(device_c, device_a, device_b);
    hipDeviceSynchronize();

    int *c;
    c = (int*)malloc(sizeof(int) * arraysize);
    hipMemcpy(c, device_c, sizeof(int) * arraysize, hipMemcpyDeviceToHost);
    printf("\n\nResult: ");
    PrintOutput(c, arraysize);

    hipFree(device_c);
    free(c);    
    hipDeviceReset();
    return 0;
}