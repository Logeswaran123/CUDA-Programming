#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "host_utils.h"
#include "cuda_utils.cuh"

#include <iostream>
#include <stdlib.h>

using namespace std;

__global__ void Transpose1(int *matrix, int *transposed, int num_cols, int num_rows) {
    // Read row
    // Write column
    // Row major traversal in input matrix

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < num_cols && iy < num_rows) {
        transposed[ix * num_rows + iy] = matrix[iy * num_cols + ix];
    }
}

__global__ void Transpose2(int *matrix, int *transposed, int num_cols, int num_rows) {
    // Read column
    // Write row
    // Column major traversal in input matrix

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < num_cols && iy < num_rows) {
        transposed[iy * num_cols + ix] = matrix[ix * num_rows + iy];
    }
}

__global__ void Transpose1_Unroll(int *matrix, int *transposed, int num_cols, int num_rows) {
    // Read row
    // Write column
    // Row major traversal in input matrix
    // Unroll Loop

    int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int ti = iy * num_cols + ix;
    int to = ix * num_rows + iy;

    if (ix + 3 * blockDim.x < num_cols && iy < num_rows) {
        transposed[to + num_rows * 0 * blockDim.x] = matrix[ti + 0 * blockDim.x];
        transposed[to + num_rows * 1 * blockDim.x] = matrix[ti + 1 * blockDim.x];
        transposed[to + num_rows * 2 * blockDim.x] = matrix[ti + 2 * blockDim.x];
        transposed[to + num_rows * 3 * blockDim.x] = matrix[ti + 3 * blockDim.x];
    }
}

__global__ void Transpose2_Unroll(int *matrix, int *transposed, int num_cols, int num_rows) {
    // Read column
    // Write row
    // Column major traversal in input matrix
    // Unroll Loop

    int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int ti = iy * num_cols + ix;
    int to = ix * num_rows + iy;

    if (ix + 3 * blockDim.x < num_cols && iy < num_rows) {
        transposed[ti + 0 * blockDim.x] = matrix[to + 0 * blockDim.x*num_rows];
        transposed[ti + 1 * blockDim.x] = matrix[to + 1 * blockDim.x*num_rows];
        transposed[ti + 2 * blockDim.x] = matrix[to + 2 * blockDim.x*num_rows];
        transposed[ti + 3 * blockDim.x] = matrix[to + 3 * blockDim.x*num_rows];
    }
}

__global__ void TransposeDiagonal(int *matrix, int *transposed, int num_cols, int num_rows) {
    int blk_x = blockIdx.x;
    int blk_y = (blockIdx.x + blockIdx.y) % gridDim.x;

    int ix = blockIdx.x * blk_x + threadIdx.x;
    int iy = blockIdx.y * blk_y + threadIdx.y;

    if (ix < num_cols && iy < num_rows) {
        transposed[ix * num_rows + iy] = matrix[iy * num_cols + ix];
    }
}

int main() {
    cout << "\n---------------------Matrix Transpose---------------------\n" << endl;
    clock_t cpu_start, cpu_end;

    int num_rows = 1024;
    int num_cols = 1024;
    int size = num_rows * num_cols; // 1MB
    int byte_size = size * sizeof(int);

    int *host_input, *host_transposed, *device_result;
    host_input = (int*)malloc(byte_size);
    host_transposed = (int*)malloc(byte_size);

    InitializeData(host_input, size, INIT_RANGE);

    // Perform matrix transpose on CPU
    cout << "\n-----Transpose on CPU-----" << endl;
    cpu_start = clock();
    MatrixTranposeCPU(host_input, host_transposed, num_cols, num_rows);
    cpu_end = clock();

    printf("\nCPU execution time (Function Only): %4.6f milliseconds\n",
        (double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC) * 1000.0);

    /***************************************************************************/
    float kernel_milliseconds = 0;
    int block_x = 128;
    int block_y = 8;
    hipEvent_t kernel_start, kernel_end;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_end);

    int *device_matrix, *device_transposed;
    GPUErrorCheck(hipMalloc((void**)&device_matrix, byte_size));
    GPUErrorCheck(hipMemcpy(device_matrix, host_input, byte_size, hipMemcpyHostToDevice));

    dim3 blocks(block_x, block_y);
    dim3 grid(num_cols / block_x + 1, num_rows / block_y + 1);

    // Transpose: Read Row, Write Column
    cout << "\n-----Transpose on Device: Read Row, Write Column-----" << endl;
    GPUErrorCheck(hipMalloc((void**)&device_transposed, byte_size));
    GPUErrorCheck(hipEventRecord(kernel_start, 0));
    Transpose1<<<grid, blocks>>>(device_matrix, device_transposed, num_cols, num_rows);
    GPUErrorCheck(hipEventRecord(kernel_end, 0));
    GPUErrorCheck(hipEventSynchronize(kernel_end));
    GPUErrorCheck(hipDeviceSynchronize());

    device_result = (int*)malloc(byte_size);
    GPUErrorCheck(hipMemcpy(device_result, device_transposed, byte_size, hipMemcpyDeviceToHost));
    CompareArrays(host_transposed, device_result, size);
    GPUErrorCheck(hipFree(device_transposed));
    free(device_result);

    GPUErrorCheck(hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_end));
    printf("GPU Execution Time (Kernel Only): %4.6f milliseconds\n", kernel_milliseconds);

    // Transpose: Read Column, Write Row
    cout << "\n-----Transpose on Device: Read Column, Write Row-----" << endl;
    GPUErrorCheck(hipMalloc((void**)&device_transposed, byte_size));
    GPUErrorCheck(hipEventRecord(kernel_start, 0));
    Transpose2<<<grid, blocks>>>(device_matrix, device_transposed, num_cols, num_rows);
    GPUErrorCheck(hipEventRecord(kernel_end, 0));
    GPUErrorCheck(hipEventSynchronize(kernel_end));
    GPUErrorCheck(hipDeviceSynchronize());

    device_result = (int*)malloc(byte_size);
    GPUErrorCheck(hipMemcpy(device_result, device_transposed, byte_size, hipMemcpyDeviceToHost));
    CompareArrays(host_transposed, device_result, size);
    GPUErrorCheck(hipFree(device_transposed));
    free(device_result);

    GPUErrorCheck(hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_end));
    printf("GPU Execution Time (Kernel Only): %4.6f milliseconds\n", kernel_milliseconds);

    // Transpose: Read Row, Write Column, with Unrolled loop
    cout << "\n-----Transpose on Device: Read Row, Write Column, with Unrolled loop-----" << endl;
    GPUErrorCheck(hipMalloc((void**)&device_transposed, byte_size));
    GPUErrorCheck(hipEventRecord(kernel_start, 0));
    Transpose1_Unroll<<<grid, blocks>>>(device_matrix, device_transposed, num_cols, num_rows);
    GPUErrorCheck(hipEventRecord(kernel_end, 0));
    GPUErrorCheck(hipEventSynchronize(kernel_end));
    GPUErrorCheck(hipDeviceSynchronize());

    device_result = (int*)malloc(byte_size);
    GPUErrorCheck(hipMemcpy(device_result, device_transposed, byte_size, hipMemcpyDeviceToHost));
    CompareArrays(host_transposed, device_result, size);
    GPUErrorCheck(hipFree(device_transposed));
    free(device_result);

    GPUErrorCheck(hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_end));
    printf("GPU Execution Time (Kernel Only): %4.6f milliseconds\n", kernel_milliseconds);

    // Transpose: Read Column, Write Row, with Unrolled loop
    cout << "\n-----Transpose on Device: Read Column, Write Row, with Unrolled loop-----" << endl;
    GPUErrorCheck(hipMalloc((void**)&device_transposed, byte_size));
    GPUErrorCheck(hipEventRecord(kernel_start, 0));
    Transpose2_Unroll<<<grid, blocks>>>(device_matrix, device_transposed, num_cols, num_rows);
    GPUErrorCheck(hipEventRecord(kernel_end, 0));
    GPUErrorCheck(hipEventSynchronize(kernel_end));
    GPUErrorCheck(hipDeviceSynchronize());

    device_result = (int*)malloc(byte_size);
    GPUErrorCheck(hipMemcpy(device_result, device_transposed, byte_size, hipMemcpyDeviceToHost));
    CompareArrays(host_transposed, device_result, size);
    GPUErrorCheck(hipFree(device_transposed));
    free(device_result);

    GPUErrorCheck(hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_end));
    printf("GPU Execution Time (Kernel Only): %4.6f milliseconds\n", kernel_milliseconds);

    // // Transpose: Diagonal method
    // cout << "\n-----Transpose on Device: Diagonal method-----" << endl;
    // GPUErrorCheck(hipMalloc((void**)&device_transposed, byte_size));
    // GPUErrorCheck(hipEventRecord(kernel_start, 0));
    // TransposeDiagonal<<<grid, blocks>>>(device_matrix, device_transposed, num_cols, num_rows);
    // GPUErrorCheck(hipEventRecord(kernel_end, 0));
    // GPUErrorCheck(hipEventSynchronize(kernel_end));
    // GPUErrorCheck(hipDeviceSynchronize());

    // device_result = (int*)malloc(byte_size);
    // GPUErrorCheck(hipMemcpy(device_result, device_transposed, byte_size, hipMemcpyDeviceToHost));
    // CompareArrays(host_transposed, device_result, size);
    // GPUErrorCheck(hipFree(device_transposed));
    // free(device_result);

    // GPUErrorCheck(hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_end));
    // printf("GPU Execution Time (Kernel Only): %4.6f milliseconds\n", kernel_milliseconds);

    GPUErrorCheck(hipFree(device_matrix));
    free(host_transposed);
    free(host_input);

    GPUErrorCheck(hipDeviceReset());
    return 0;
}